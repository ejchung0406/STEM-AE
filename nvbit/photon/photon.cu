#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2019 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <pthread.h>
#include <stdint.h>
#include <stdio.h>
#include <unordered_set>

#include <random>
#include <cmath>
#include <fstream>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* nvbit utility functions */
#include "utils/utils.h"

/* kernel id counter, maintained in system memory */
uint32_t kernel_id = 0;

/* total instruction counter, maintained in system memory, incremented by
 * "counter" every time a kernel completes  */
uint64_t tot_app_instrs = 0;

/* kernel instruction counter, updated by the GPU threads */
__managed__ uint64_t counter = 0;
__managed__ uint64_t counter_pred_off = 0;
__managed__ uint64_t **ptr_to_bb_counters = nullptr;  // Pointer to the basic block counters
__managed__ uint64_t *bb_counters = nullptr;  // Flattened array: [bb_idx * num_warps + warp_id]
// __managed__ uint64_t *function_offsets = nullptr;  // Array to store starting index for each function's counters

/* global control variables for this tool */
uint32_t start_grid_num = 0;
uint32_t end_grid_num = UINT32_MAX;
int verbose = 0;
int count_warp_level = 1;
int exclude_pred_off = 0;
int active_from_start = 1;
bool mangled = false;

/* used to select region of insterest when active from start is off */
bool active_region = true;

/* a pthread mutex, used to prevent multiple kernels to run concurrently and
 * therefore to "corrupt" the counter variable */
pthread_mutex_t mutex;

/* Path to save BBV csv file */
std::string bbv_file_path = "./basic_block_counters.csv";

/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We
 * typically do initializations in this call. In this case for instance we get
 * some environment variables values which we use as input arguments to the tool
 */
void nvbit_at_init() {
    /* just make sure all managed variables are allocated on GPU */
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);

    /* we get some environment variables that are going to be use to selectively
     * instrument (within a interval of kernel indexes and instructions). By
     * default we instrument everything. */
    GET_VAR_INT(start_grid_num, "START_GRID_NUM", 0,
                "Beginning of the kernel gird launch interval where to apply "
                "instrumentation");
    GET_VAR_INT(end_grid_num, "END_GRID_NUM", UINT32_MAX,
                "End of the kernel grid launch interval where to apply "
                "instrumentation");
    GET_VAR_INT(count_warp_level, "COUNT_WARP_LEVEL", 1,
                "Count warp level or thread level instructions");
    GET_VAR_INT(exclude_pred_off, "EXCLUDE_PRED_OFF", 0,
                "Exclude predicated off instruction from count");
    GET_VAR_INT(
        active_from_start, "ACTIVE_FROM_START", 1,
        "Start instruction counting from start or wait for hipProfilerStart "
        "and hipProfilerStop");
    GET_VAR_INT(mangled, "MANGLED_NAMES", 1,
                "Print kernel names mangled or not");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    GET_VAR_STR(bbv_file_path, "BBV_FILE_PATH",
                "Path to save the basic block counters CSV file");
    if (active_from_start == 0) {
        active_region = false;
    }

    std::string pad(100, '-');
    printf("%s\n", pad.c_str());

    /* set mutex as recursive */
    pthread_mutexattr_t attr;
    pthread_mutexattr_init(&attr);
    pthread_mutexattr_settype(&attr, PTHREAD_MUTEX_RECURSIVE);
    pthread_mutex_init(&mutex, &attr);
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    uint64_t function_id = 0;
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }

        /* Get the static control flow graph of instruction */
        const CFG_t &cfg = nvbit_get_CFG(ctx, f);
        if (cfg.is_degenerate) {
            printf(
                "Warning: Function %s is degenerated, we can't compute basic "
                "blocks statically",
                nvbit_get_func_name(ctx, f));
        }

        if (verbose) {
            printf("Function %s\n", nvbit_get_func_name(ctx, f));
            /* print */
            uint64_t cnt = 0;
            for (auto &bb : cfg.bbs) {
                printf("Basic block id %lu - num instructions %ld\n", cnt++,
                       bb->instrs.size());
                for (auto &i : bb->instrs) {
                    i->print(" ");
                }
            }
        }

        if (verbose) {
            printf("inspecting %s - number basic blocks %ld\n",
                   nvbit_get_func_name(ctx, f), cfg.bbs.size());
        }

        /* Iterate on basic block and inject the first instruction */
        uint64_t bb_id = 0;
        for (auto &bb : cfg.bbs) {
            Instr *i = bb->instrs[0];

            uint64_t counter_index = bb_id;
            
            /* inject device function */
            nvbit_insert_call(i, "count_bbv", IPOINT_BEFORE);
            /* add count warp level option */
            nvbit_add_call_arg_const_val32(i, count_warp_level);
            /* BB id */
            nvbit_add_call_arg_const_val64(i, counter_index);
            /* add pointer to counter location */
            nvbit_add_call_arg_const_val64(i, (uint64_t)&ptr_to_bb_counters);
            if (verbose) {
                printf("Injecting count_bbv for Function %lu BB %lu with counter at %p, %p, %p\n", 
                       function_id, bb_id, ptr_to_bb_counters, *ptr_to_bb_counters, &bb_counters[counter_index]);
                i->print("Inject count_instr before - ");

                unsigned long long* bbc = *(unsigned long long**)(uint64_t)&ptr_to_bb_counters;
                printf("%p, %p\n", bbc, bbc+bb_id);
            }

            // uint64_t counter_index = (function_offsets[function_id] + bb_id) * num_of_warps;
            
            // /* inject device function */
            // nvbit_insert_call(i, "count_bbv_per_warp", IPOINT_BEFORE);
            // /* add count warp level option */
            // nvbit_add_call_arg_const_val32(i, count_warp_level);
            // /* add pointer to counter location */
            // nvbit_add_call_arg_const_val64(i, (uint64_t)&bb_counters[counter_index]);
            // if (verbose) {
            //     printf("Injecting count_bbv for Function %lu BB %lu with counter at %p\n", 
            //            function_id, bb_id, &bb_counters[counter_index]);
            //     i->print("Inject count_instr before - ");
            // }
            bb_id++;
        }
        function_id++;
    }
}

std::vector<uint64_t> warp_sampling(uint64_t num_of_warps, double chance) {
    uint64_t sample_size = std::ceil(num_of_warps * chance);  // Compute ceil(n/100)
    std::unordered_set<uint64_t> unique_samples;  // Set to ensure no duplicates
    
    // Random number generator setup
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, num_of_warps - 1);

    // Keep generating random numbers until we have the desired number of unique samples
    while (unique_samples.size() < sample_size) {
        unique_samples.insert((uint64_t)dis(gen));  // Insert random number into set
    }

    // Convert set to vector for return
    std::vector<uint64_t> sampled_numbers(unique_samples.begin(), unique_samples.end());
    
    return sampled_numbers;
}

/* This call-back is triggered every time a CUDA driver call is encountered.
 * Here we can look for a particular CUDA driver call by checking at the
 * call back ids  which are defined in tools_cuda_api_meta.h.
 * This call back is triggered bith at entry and at exit of each CUDA driver
 * call, is_exit=0 is entry, is_exit=1 is exit.
 * */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    
    
    /* Identify all the possible CUDA launch events */
    if (cbid == API_CUDA_cuLaunch || cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchGrid || cbid == API_CUDA_cuLaunchGridAsync ||
        cbid == API_CUDA_cuLaunchKernel ||
        cbid == API_CUDA_cuLaunchKernelEx ||
        cbid == API_CUDA_cuLaunchKernelEx_ptsz) {
        /* cast params to launch parameter based on cbid since if we are here
         * we know these are the right parameters types */
        hipFunction_t func;
        uint64_t num_of_warps;
        if (cbid == API_CUDA_cuLaunchKernelEx_ptsz ||
            cbid == API_CUDA_cuLaunchKernelEx) {
            cuLaunchKernelEx_params* p = (cuLaunchKernelEx_params*)params;
            func = p->f;
            // Calculate total number of warps
            // Each block has (blockDimX * blockDimY * blockDimZ) / 32 warps
            uint64_t threads_per_block = p->config->blockDimX * p->config->blockDimY * p->config->blockDimZ;
            uint64_t warps_per_block = (threads_per_block + 31) / 32;  // Round up division
            num_of_warps = warps_per_block * p->config->gridDimX * p->config->gridDimY * p->config->gridDimZ;
        } else {
            cuLaunchKernel_params* p = (cuLaunchKernel_params*)params;
            func = p->f;
            // Calculate total number of warps
            uint64_t threads_per_block = p->blockDimX * p->blockDimY * p->blockDimZ;
            uint64_t warps_per_block = (threads_per_block + 31) / 32;  // Round up division
            num_of_warps = warps_per_block * p->gridDimX * p->gridDimY * p->gridDimZ;
        }

        if (!is_exit) {
            /* if we are entering in a kernel launch:
             * 1. Lock the mutex to prevent multiple kernels to run concurrently
             * (overriding the counter) in case the user application does that
             * 2. Instrument the function if needed
             * 3. Select if we want to run the instrumented or original
             * version of the kernel
             * 4. Reset the kernel instruction counter */

            pthread_mutex_lock(&mutex);

            // First pass: count total basic blocks and allocate memory if needed
            uint64_t total_basic_blocks = 0;

            std::vector<hipFunction_t> related_functions = nvbit_get_related_functions(ctx, func);
            related_functions.push_back(func);
            uint64_t num_functions = related_functions.size();
            for (auto f : related_functions) {
                const CFG_t &cfg = nvbit_get_CFG(ctx, f);
                total_basic_blocks += cfg.bbs.size();
            }

            // Allocate memory for counters (flattened) and offsets
            // uint64_t total_counters = total_basic_blocks * num_of_warps;
            uint64_t total_counters = total_basic_blocks;
            
            // CUDA_SAFECALL(hipMallocManaged(&bb_counters, total_counters * sizeof(uint64_t), hipMemAttachGlobal));
            CUDA_SAFECALL(hipMallocManaged(&bb_counters, total_counters * sizeof(uint64_t), hipMemAttachHost));
            CUDA_SAFECALL(hipMemset(bb_counters, 0, total_counters * sizeof(uint64_t)));

            ptr_to_bb_counters = &bb_counters;
            
            CUDA_SAFECALL(hipDeviceSynchronize());
            instrument_function_if_needed(ctx, func);

            if (active_from_start) {
                if (kernel_id >= start_grid_num && kernel_id < end_grid_num) {
                    active_region = true;
                } else {
                    active_region = false;
                }
            }

            if (active_region) {
                nvbit_enable_instrumented(ctx, func, true);
            } else {
                nvbit_enable_instrumented(ctx, func, false);
            }

            counter = 0;
            counter_pred_off = 0;

        } else {
            /* if we are exiting a kernel launch:
             * 1. Wait until the kernel is completed using
             * hipDeviceSynchronize()
             * 2. Get number of thread blocks in the kernel
             * 3. Print the thread instruction counters
             * 4. Release the lock*/
            CUDA_SAFECALL(hipDeviceSynchronize());
            uint64_t kernel_instrs = counter - counter_pred_off;
            tot_app_instrs += kernel_instrs;
            int num_ctas = 0;
            uint64_t num_threads = 0;
            if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
                cbid == API_CUDA_cuLaunchKernel) {
                cuLaunchKernel_params *p2 = (cuLaunchKernel_params *)params;
                num_ctas = p2->gridDimX * p2->gridDimY * p2->gridDimZ;
                num_threads = num_ctas * p2->blockDimX * p2->blockDimY * p2->blockDimZ;
            } else if (cbid == API_CUDA_cuLaunchKernelEx_ptsz ||
                cbid == API_CUDA_cuLaunchKernelEx) {
                cuLaunchKernelEx_params *p2 = (cuLaunchKernelEx_params *)params;
                num_ctas = p2->config->gridDimX * p2->config->gridDimY *
                    p2->config->gridDimZ;
                num_threads = num_ctas * p2->config->blockDimX * p2->config->blockDimY * p2->config->blockDimZ;
            }
            printf(
                "kernel %d - %s - #thread-blocks %d, #warps %lu, #threads %lu\n", 
                kernel_id, nvbit_get_func_name(ctx, func, mangled), num_ctas, 
                num_of_warps, num_threads);

            // Print final values of basic block counters
            if (bb_counters != nullptr) {
                std::vector<uint64_t> sampled_warps = warp_sampling(num_of_warps, 0.01);
                // if (verbose) {
                //     printf("\nBasic Block Counters for kernel %d:\n", kernel_id);
                //     for (const w: sampled_warps) {
                //         printf("  BBV for warp %ld: ", (uint64_t)w);
                //         for (uint64_t bb = 0; bb < total_basic_blocks; bb++) {
                //             printf("%ld ", bb_counters[bb * num_of_warps + w]);
                //         }
                //         printf("\n");
                //     }
                // }

                std::ofstream ofs(bbv_file_path, std::ios::app);
                if (!ofs.is_open()) {
                    std::cerr << "Failed to open basic_block_counters.csv" << std::endl;
                    exit(EXIT_FAILURE);
                }

                std::vector<hipFunction_t> related_functions = nvbit_get_related_functions(ctx, func);
                related_functions.push_back(func);
                uint64_t total_basic_blocks = 0;
                uint64_t num_functions = related_functions.size();
                for (auto f : related_functions) {
                    const CFG_t &cfg = nvbit_get_CFG(ctx, f);
                    total_basic_blocks += cfg.bbs.size();
                }

                if (kernel_id == 0)
                    ofs << "Warp ID," << "Basic Block Counts" << std::endl;
                ofs << "Kernel" << kernel_id << "," << total_basic_blocks << "," << num_of_warps << "," << nvbit_get_func_name(ctx, func, mangled) << std::endl;

                for (uint64_t bb = 0; bb < total_basic_blocks; bb++) {
                    ofs << bb_counters[bb] << ",";
                }
                ofs << std::endl;    

                // for (const w: sampled_warps) {
                //     ofs << w << ",";
                //     for (uint64_t bb = 0; bb < total_basic_blocks; bb++) {
                //         ofs << bb_counters[bb * num_of_warps + w] << ",";
                //     }
                //     ofs << std::endl;
                // }
                ofs.close();

                // Free the allocated memory using CUDA free
                CUDA_SAFECALL(hipFree(bb_counters));
                // CUDA_SAFECALL(hipFree(function_offsets));
                bb_counters = nullptr;
                // function_offsets = nullptr;
                total_basic_blocks = 0;
            }

            pthread_mutex_unlock(&mutex);
            kernel_id++;
        }
    } else if (cbid == API_CUDA_cuProfilerStart && is_exit) {
        if (!active_from_start) {
            active_region = true;
        }
    } else if (cbid == API_CUDA_cuProfilerStop && is_exit) {
        if (!active_from_start) {
            active_region = false;
        }
    }
    
}

void nvbit_at_term() {
    // printf("Total app instructions: %ld\n", tot_app_instrs);
}
