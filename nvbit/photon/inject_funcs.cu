#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2019 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdint.h>
#include <stdio.h>

#include "utils/utils.h"

__inline__ __device__ uint64_t get_flat_wid() {
	uint64_t bid = blockIdx.x + (gridDim.x * (blockIdx.y + (blockIdx.z * gridDim.y))); // block id 
    int warpSize = 32;
    uint64_t warps_per_block = (blockDim.x * blockDim.y * blockDim.z + warpSize - 1) / warpSize;
	return get_warpid() + bid * warps_per_block;
}

extern "C" __device__ __noinline__ void count_bbv(int count_warp_level,
                                                    uint64_t bid,
                                                    uint64_t pcounter) {
    /* all the active threads will compute the active mask */
    const int active_mask = __ballot_sync(__activemask(), 1);

    /* each thread will get a lane id (get_lane_id is implemented in
    * utils/utils.h) */
    const int laneid = get_laneid();

    /* get the id of the first active thread */
    const int first_laneid = __ffs(active_mask) - 1;

    /* count all the active thread */
    const int num_threads = __popc(active_mask);

    unsigned long long* bb_counters = **(unsigned long long***)pcounter;

    /* only the first active thread will perform the atomic */
    if (first_laneid == laneid) {
        if (count_warp_level) {
            atomicAdd(bb_counters + bid, 1);
        } else {
            atomicAdd(bb_counters + bid, num_threads);
        }
    }
}

extern "C" __device__ __noinline__ void count_bbv_per_warp(int count_warp_level,
                                                     uint64_t pcounter) {
    /* all the active threads will compute the active mask */
    const int active_mask = __ballot_sync(__activemask(), 1);

    /* each thread will get a lane id (get_lane_id is implemented in
     * utils/utils.h) */
    const int laneid = get_laneid();

    /* get the id of the first active thread */
    const int first_laneid = __ffs(active_mask) - 1;

    /* count all the active thread */
    const int num_threads = __popc(active_mask);

    const uint64_t wid = get_global_warp_id();

    /* only the first active thread will perform the atomic */
    if (first_laneid == laneid) {
        if (count_warp_level) {
            atomicAdd((unsigned long long*)pcounter + wid, 1);
        } else {
            atomicAdd((unsigned long long*)pcounter + wid, num_threads);
        }
    }
}

extern "C" __device__ __noinline__ void count_pred_off(int predicate,
                                                       int count_warp_level,
                                                       uint64_t pcounter) {
    /* all the active threads will compute the active mask */
    const int active_mask = __ballot_sync(__activemask(), 1);

    /* each thread will get a lane id (get_lane_id is implemented in
     * utils/utils.h) */
    const int laneid = get_laneid();

    /* get the id of the first active thread */
    const int first_laneid = __ffs(active_mask) - 1;

    /* get predicate mask */
    const int predicate_mask = __ballot_sync(__activemask(), predicate);

    /* get mask of threads that have their predicate off */
    const int mask_off = active_mask ^ predicate_mask;

    /* count the number of threads that have their predicate off */
    const int num_threads_off = __popc(mask_off);

    /* only the first active thread updates the counter of predicated off
     * threads */
    if (first_laneid == laneid) {
        if (count_warp_level) {
            if (predicate_mask == 0) {
                atomicAdd((unsigned long long*)pcounter, 1);
            }
        } else {
            atomicAdd((unsigned long long*)pcounter, num_threads_off);
        }
    }
}
